
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#define N 9000000

#define THDS_PER_BLK 256
__global__ void sum_reduce(double *arr, double *sum)
{
    int myid = blockIdx.x*blockDim.x + threadIdx.x;
	double tmp_sum = 0.0;
	__shared__ double tmp[THDS_PER_BLK];
    if(myid<N)
    {
        tmp[threadIdx.x] = arr[myid];
        __syncthreads();
        if(threadIdx.x == 0)
        {
            for(int i=0;i<THDS_PER_BLK;i++)
            {
                tmp_sum += tmp[threadIdx.x]; 
            }
            sum[blockIdx.x] = tmp_sum;
        }
    }
}

__global__ void pi_calc(double *sum)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;	
	double x, step;
	if(myid<N)
	{
	    step = 1.0/(double)N;
        x = (myid)*step;
        sum[myid] = 4.0/(1.0+x*x);
    }
}
/*
step = 1.0/(double)N;
        for(i=0; i<N; i++){
                x = (i)*step;
                sum = sum + 4.0/(1.0+x*x);
        }
        pi = step*sum;
*/
int main()
{
	double *sum, *sum_d, *sum_small_d;
	int i=0;
	double total = 0.0;
	double pi, step;
	double exe_time;
	step = 1.0/(double)N;
	struct timeval stop_time, start_time;
    
    int thds_per_block = THDS_PER_BLK;
	int num_blocks = (N/thds_per_block)+1;
	
	sum = (double *)malloc(N*sizeof(double));
	
	hipMalloc(&sum_d, N*sizeof(double));
	hipMalloc(&sum_small_d, num_blocks*sizeof(double));
	
	gettimeofday(&start_time, NULL);
	
	
	pi_calc<<< num_blocks,thds_per_block >>>(sum_d);
	hipDeviceSynchronize();
	
	sum_reduce<<< num_blocks,thds_per_block >>>(sum_d, sum_small_d);
	hipMemcpy(sum, sum_small_d, num_blocks*sizeof(double), hipMemcpyDeviceToHost);
	
	for(i=0; i<num_blocks; i++)
	{
        total += sum[i];
    }
    pi = step*total;
    
    gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	    
    printf("\n pi = %lf and exe_time = %lf\n", pi, exe_time);	
    hipFree(sum_d); 
    free(sum);
}
