
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 100
#define THDS_PER_BLK 256

__device__ int square(int myid)
{
	int sq = myid*myid;
	return sq;
}

__global__ void calc_square(int* a_d)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;
	a_d[myid] = square(myid);
}

int main()
{
	int *a_d;
	int size = N * sizeof(int);
	int a[N];
	int i=0;
	
    	int thds_per_block = THDS_PER_BLK;
	int num_blocks = (N/thds_per_block)+1;
	
	//Initialize the vectors
	for(i=0; i<N; i++ )
	{
		a[i] = i;
	}
	
	hipMalloc(&a_d, size);
	hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);	
	
	calc_square<<< num_blocks,thds_per_block >>>(a_d); 
	
	hipMemcpy(a, a_d, N*sizeof(int), hipMemcpyDeviceToHost);


	// print the output
	for(i=0; i<N; i++ )
	{
		printf("\t%d",a[i]);
	}
	
	hipFree(a_d);	
}




