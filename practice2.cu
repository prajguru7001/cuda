
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 10
#define THDS_PER_BLK 256


__device__ double calc(double a, double b)
{
	double alpha = 0.001;
	double ans = a + alpha*b;
	return ans;
}

__global__ void calc_square(double* a_d, double* b_d, double* c_d)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;
	
	c_d[myid] = calc(a_d[myid], b_d[myid]);
}

int main()
{
	double *a_d, *b_d, *c_d;
	int size = N * sizeof(double);
	double a[N], b[N], c[N];
	int i=0;
	
    	int thds_per_block = THDS_PER_BLK;
	int num_blocks = (N/thds_per_block)+1;
	
	//Initialize the vectors
	for(i=0; i<N; i++ )
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}
	
	hipMalloc(&a_d, size);
	hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);	
	
	hipMalloc(&b_d, size);
	hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);
	
	hipMalloc(&c_d, size);
	
	calc_square<<< num_blocks,thds_per_block >>>(a_d, b_d, c_d); 
	
	hipMemcpy(c, c_d, size, hipMemcpyDeviceToHost);


	// print the output
	for(i=0; i<N; i++ )
	{
		printf("\t%lf",c[i]);
	}
	
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);	
}




